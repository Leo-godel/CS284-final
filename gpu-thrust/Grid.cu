#include "hip/hip_runtime.h"
#include "Grid.h"
#include <cassert>
#include <cstdio>

#define MATRIX_EPSILON 1e-6

__host__ __device__ double bspline(double x) {
	x = fabs(x);
	double w;
	if (x < 1)
		w = x * x * (x / 2 - 1) + 2 / 3.0;
	else if (x < 2)
		w = x * (x * (-x / 6 + 1) - 2) + 4 / 3.0;
	else return 0;

	return w;
}
//Slope of interpolation function
__host__ __device__ double bsplineSlope(double x) {
	double abs_x = fabs(x);
	if (abs_x < 1)
		return 1.5 * x * abs_x - 2 * x;
	else if (x < 2)
		return -x * abs_x / 2 + 2 * x - 2 * x / abs_x;
	else return 0;
}

__host__ __device__ Matrix2D outer_product(Vector2D& a, Vector2D& b) {
	return Matrix2D(a.x * b.x, a.x * b.y, a.y * b.x, a.y * b.y);
}

__host__ __device__ void polarDecomp(Matrix2D m, Matrix2D& R, Matrix2D& S) {
	auto x = m(0, 0) + m(1, 1);
	auto y = m(1, 0) - m(0, 1);
	auto scale = 1.0 / sqrt(x * x + y * y);
	auto c = x * scale, s = y * scale;
	R(0, 0) = c;
	R(0, 1) = -s;
	R(1, 0) = s;
	R(1, 1) = c;
	S = R.T() * m;
}

__host__ __device__ void svd(Matrix2D m, Matrix2D& U, Matrix2D& sig, Matrix2D& V){
    if (fabs(m(1, 0) - m(0, 1)) < MATRIX_EPSILON && fabs(m(1, 0)) < MATRIX_EPSILON) {
        U = Matrix2D(m(0, 0) < 0 ? -1 : 1, 0, 0, m(1, 1) < 0 ? -1 : 1);
        sig(0, 0) = fabs(m(0, 0)), sig(1, 1) = fabs(m(1, 1));
        V = Matrix2D();
    }
    else {
        double j = m(0, 0) * m(0, 0) + m(1, 0) * m(1, 0);
        double k = m(0, 1) * m(0, 1) + m(1, 1) * m(1, 1);
        double v_c = m(0, 0) * m(0, 1) + m(1, 0) * m(1, 1);

        if (fabs(v_c) < MATRIX_EPSILON) {
            double s1 = sqrt(j);
            double s2 = fabs(j - k) < MATRIX_EPSILON ? s1 : sqrt(k);
            sig(0, 0) = s1, sig(1, 1) = s2;
            V = Matrix2D();
            U = Matrix2D(m(0, 0) / s1, m(0, 1) / s2, m(1, 0) / s1, m(1, 1) / s2);
        }
        else {
            double jmk = j - k,
                    jpk = j + k,
                    root = sqrt(jmk * jmk + 4 * v_c * v_c),
                    eig = (jpk + root) / 2,
                    s1 = sqrt(eig),
                    s2 = fabs(root) < MATRIX_EPSILON ? s1 : sqrt((jpk - root) / 2);
            sig(0, 0) = s1, sig(1, 1) = s2;
            double v_s = eig - j,
                    len = sqrt(v_s * v_s + v_c * v_c);
            v_c /= len;
            v_s /= len;
            V = Matrix2D(v_c, -v_s, v_s, v_c);
            U = Matrix2D(
                    (m(0, 0) * v_c + m(0, 1) * v_s) / s1,
                    (m(0, 1) * v_c - m(0, 0) * v_s) / s2,
                    (m(1, 0) * v_c + m(1, 1) * v_s) / s1,
                    (m(1, 1) * v_c - m(1, 0) * v_s) / s2
            );
        }
    }
}

__device__ double my_atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

struct OBCmp {
	__host__ __device__
	bool operator()(const SpGrid& o1, const SpGrid& o2) {
		return o1.node_id< o2.node_id;
	}
};

__host__ void Grid::initGridMassVel() {
	// Map particle to grid
	Node* grid_ptr = thrust::raw_pointer_cast(&nodes[0]);

	auto func = [=] __device__ (Particle & p) {
		Vector2D _origin(0, 0), _node_size(1. / 128., 1. /128.), _size(256 + 1, 128 + 1);
		// get the index of the grid cross point corresponding to the particle (it is on the bottom left of the particle)
		p.grid_p = (p.pos - _origin) / _node_size;
		int p_x = (int)p.grid_p.x;  // x coord index in grid
		int p_y = (int)p.grid_p.y;  // y coord index in grid

		// Map from (p_x - 1, p_y - 1) to (p_x + 2, p_y + 2)
		// The origin is bottom left, which means node_id = y * size.x + x
		for (int it = 0, y = p_y - 1; y <= p_y + 2; ++y) {
			if (y < 0 || y >= _size.y) // here size.y has already been added by 1
				continue;
			// Y interpolation
			double weight_y = bspline(p.grid_p.y - y);
			double dy = bsplineSlope(p.grid_p.y - y);

			for (int x = p_x - 1; x <= p_x + 2; ++x, ++it) {
				if (x < 0 || x >= _size.x)
					continue;

				// X interpolation
				double weight_x = bspline(p.grid_p.x - x);
				double dx = bsplineSlope(p.grid_p.x - x);

				// set weight of particles related nodes
				double w = weight_x * weight_y;
				p.weights[it] = w;

				// set weight gradient
				p.weight_gradient[it] = Vector2D(dx * weight_y, dy * weight_x);
				p.weight_gradient[it] /= _node_size;

				// set node weighted mass and velocity
				int node_id = int(y * _size.x + x);

				//nodes[node_id].mass += w * p.mass;
				//my_atomicAdd(&(grid_ptr[node_id].mass), w * p.mass);
				//nodes[node_id].vel += p.vel * w * p.mass;
				Vector2D temp = p.vel * w * p.mass;
				//my_atomicAdd(&(grid_ptr[node_id].vel.x), temp.x);
				//my_atomicAdd(&(grid_ptr[node_id].vel.y), temp.y);
				//nodes[node_id].active = true;
				//atomicAdd(&(grid_ptr[node_id].active), 1);
			}
		}
	};
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func);
	//cout << other_particles.size() << endl;
	thrust::sort(thrust::device, other_particles.begin(), other_particles.end(), OBCmp());

	thrust::for_each(
		thrust::device,
		nodes.begin(),
		nodes.end(),
		[=] __device__(Node& n) {
			if (n.active)
				n.vel /= n.mass;
		}
	);

}

// Calculate particles'volumes
__host__ void Grid::initVolumes() {
	Node* grid_ptr = thrust::raw_pointer_cast(&nodes[0]);

	auto func = [=] __device__(Particle & p) {
		Vector2D _origin(0, 0), _node_size(1. / 128., 1. /128.), _size(256 + 1, 128 + 1);
		double _node_area = _node_size.x * _node_size.y;

		int p_x = (int)p.grid_p.x;
		int p_y = (int)p.grid_p.y;

		p.density = 0;
		for (int it = 0, y = p_y - 1; y <= p_y + 2; ++y) {
			for (int x = p_x - 1; x <= p_x + 2; ++x, ++it) {
				if (y < 0 || y >= _size.y || x < 0 || x >= _size.x)
					continue;
				
				double w = p.weights[it];
				int node_id = int(y * _size.x + x);
				if (w > BSPLINE_EPSILON) {
					p.density += w * grid_ptr[node_id].mass;
				}
			}
		}

		p.density /= _node_area;
		p.volume = p.mass / p.density;
	};
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func);
}

//Calculate grid's velocity of next timestep
__host__ void Grid::computeForce() {
	Node* grid_ptr = thrust::raw_pointer_cast(&nodes[0]);

	auto func = [=] __device__(Particle & p) {
		// First calculate force based on mpmcourse
		Vector2D _origin(0, 0), _node_size(1. / 128., 1. /128.), _size(256 + 1, 128 + 1);
		double _node_area = _node_size.x * _node_size.y;

		Matrix2D I;
		Matrix2D U, Sig, V;

		svd(p.elastic_deformation, U, Sig, V);

		double e = std::exp(HARDENING * (1.0f - p.plastic_deformation.det()));
		double lambda = LAMBDA * e;
		double mu = MU * e;
		double Je = Sig.det();

		Matrix2D temp = (p.elastic_deformation - U * V.T()) * p.elastic_deformation.T() * 2 * mu + I * lambda * Je * (Je - 1);
		temp = temp * p.volume;

		// accumulate particle stress to grids
		int p_x = (int)p.grid_p.x;
		int p_y = (int)p.grid_p.y;
		for (int it = 0, y = p_y - 1; y <= p_y + 2; ++y) {
			for (int x = p_x - 1; x <= p_x + 2; ++x, ++it) {
				if (y < 0 || y >= _size.y || x < 0 || x >= _size.x)
					continue;

				double w = p.weights[it];
				int node_id = int(y * _size.x + x);
				
				//Node& node = nodes[node_id];
				if (w > BSPLINE_EPSILON) {
					//grid_ptr[node_id].force -= temp * p.weight_gradient[it];
					Vector2D value = temp * p.weight_gradient[it];
					//my_atomicAdd(&(grid_ptr[node_id].force.x), -value.x);
					//my_atomicAdd(&(grid_ptr[node_id].force.y), -value.y);
				}
			}
		}
	};
	thrust::sort(thrust::device, other_particles.begin(), other_particles.end(), OBCmp());
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func);
}

__host__ void Grid::updateVelocity() {
    // here is how we update grid velocity
	thrust::for_each(
		thrust::device,
		nodes.begin(),
		nodes.end(),
		[=] __device__(Node& n) {
			double timestep = 0.0001;
			Vector2D gravity(0, -9.8);

			if (n.active) {
				n.vel_new = n.vel + timestep * (gravity + n.force / n.mass);
				//printf("updated!\n");
			}
		}
	);
    collisionGrid();
}

__host__ void Grid::updateDeformation() {
	Node* grid_ptr = thrust::raw_pointer_cast(&nodes[0]);

	auto func = [=] __device__(Particle & p) {
		Vector2D _origin(0, 0), _node_size(1. / 128., 1. /128.), _size(256 + 1, 128 + 1);
		double _node_area = _node_size.x * _node_size.y;

		int p_x = (int)p.grid_p.x;
		int p_y = (int)p.grid_p.y;
		p.velocity_gradient = Matrix2D(0, 0, 0, 0);
		for (int it = 0, y = p_y - 1; y <= p_y + 2; ++y) {
			for (int x = p_x - 1; x <= p_x + 2; ++x, ++it) {
				if (y < 0 || y >= _size.y || x < 0 || x >= _size.x)
					continue;
				
				double temp = p.weights[it];
				Vector2D delta_w = p.weight_gradient[it];
				int node_id = int(y * _size.x + x);
				if (temp > BSPLINE_EPSILON) {
					p.velocity_gradient += outer_product(grid_ptr[node_id].vel_new, delta_w);
				}
			}
		}
	};
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func);

	auto func2 = [=] __device__(Particle & p) {
		Matrix2D I = Matrix2D();
		p.elastic_deformation = (I + p.velocity_gradient * TIMESTEP) * p.elastic_deformation;
		p.deformation_gradient = p.elastic_deformation * p.plastic_deformation;
		Matrix2D U, Sig, V;
		svd(p.elastic_deformation, U, Sig, V);
		for (int idx = 0; idx < 2; ++idx) {
			if (Sig(idx, idx) < CRIT_COMPRESS) {
				Sig(idx, idx) = CRIT_COMPRESS;
			}
			else if (Sig(idx, idx) > CRIT_STRETCH) {
				Sig(idx, idx) = CRIT_STRETCH;
			}
		}
		Matrix2D Sig_inv(1.0 / Sig(0, 0), 0, 0, 1.0 / Sig(1, 1));
		p.elastic_deformation = U * Sig * V.T();
		p.plastic_deformation = V * Sig_inv * U.T() * p.deformation_gradient;
	};
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func2);
}

// Map back to particles
__host__ void Grid::updateParticlesVelocity() {
	Node* grid_ptr = thrust::raw_pointer_cast(&nodes[0]);

	auto func = [=] __device__(Particle & p) {
		Vector2D _origin(0, 0), _node_size(1. / 128., 1. /128.), _size(256 + 1, 128 + 1);
		double _node_area = _node_size.x * _node_size.y;

		int p_x = (int)p.grid_p.x;
		int p_y = (int)p.grid_p.y;

		p.density = 0;

		Vector2D v_pic, v_flip = p.vel;
		for (int it = 0, y = p_y - 1; y <= p_y + 2; ++y) {
			for (int x = p_x - 1; x <= p_x + 2; ++x, ++it) {
				if (y < 0 || y >= _size.y || x < 0 || x >= _size.x)
					continue;
				
				double w = p.weights[it];
				int node_id = int(y * _size.x + x);
				if (w > BSPLINE_EPSILON) {
					//Node& node = nodes[node_id];
					v_pic += grid_ptr[node_id].vel_new * w;
					v_flip += (grid_ptr[node_id].vel_new - grid_ptr[node_id].vel) * w;
					p.density += w * grid_ptr[node_id].mass;
				}
			}
		}
		double flip_percent = .95;
		p.vel = v_flip * flip_percent + v_pic * (1 - flip_percent);
		p.density /= _node_area;
	};
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func);

	collisionParticles();
}

__host__ void Grid::updateParticlesPosition() {
	auto func = [=] __device__(Particle & p) {
		double timestep = 0.0001;
		p.pos += timestep * p.vel;
	};
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func);
}

__host__ void Grid::collisionGrid() {

	auto func = [=] __device__(Node & n) {
		Vector2D _origin(0, 0), _node_size(1. / 128., 1. /128.), _size(256 + 1, 128 + 1);
		double _node_area = _node_size.x * _node_size.y;
		double timestep = 0.0001;

		if (n.active) {
			Vector2D delta_scale = Vector2D(timestep, timestep);
			delta_scale /= _node_size;
			Vector2D new_pos = n.vel_new * delta_scale + n.pos;
			if (new_pos.x < BSPLINE_RADIUS || new_pos.x > _size.x - BSPLINE_RADIUS - 1) {
				n.vel_new.x = 0;
				n.vel_new.y *= STICKY;
			}
			if (new_pos.y < BSPLINE_RADIUS || new_pos.y > _size.y - BSPLINE_RADIUS - 1) {
				n.vel_new.x *= STICKY;
				n.vel_new.y = 0;
			}
		}
	};
	thrust::for_each(thrust::device, nodes.begin(), nodes.end(), func);

}

__host__ void Grid::collisionParticles() {
	
	auto func = [=] __device__(Particle & p) {
		Vector2D _origin(0, 0), _node_size(1. / 128., 1. /128.), _size(256 + 1, 128 + 1);
		double _node_area = _node_size.x * _node_size.y;
		double timestep = 0.0001;

		Vector2D delta_scale = Vector2D(timestep, timestep);
		delta_scale /= _node_size;

		Vector2D new_pos = p.grid_p + p.vel * delta_scale;

		if (new_pos.x < BSPLINE_RADIUS - 1 || new_pos.x > _size.x - BSPLINE_RADIUS) {
			p.vel.x *= -STICKY;
		}
		if (new_pos.y < BSPLINE_RADIUS - 1 || new_pos.y > _size.y - BSPLINE_RADIUS) {
			p.vel.y *= -STICKY;
		}
	};
	thrust::for_each(thrust::device, particles.begin(), particles.end(), func);
}
